// sequetial code with sweep

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <vector>
#include <string>
#include <ctime>
#include <iomanip>
#include <chrono>
#include <cmath>
#include <queue>
#include <climits>

#define sharedSize 1024
#define NumOfThreads 128

using namespace std;

namespace globalTimer {
    std::chrono::high_resolution_clock::time_point startTime;
    std::chrono::seconds timeLimit;
    std::chrono::high_resolution_clock::time_point endTime;

    void setLimit(std::chrono::seconds limit) {
        timeLimit = limit;
    }

    void startTimer() {
        startTime = std::chrono::high_resolution_clock::now();
    }

    template <class Duration = std::chrono::nanoseconds>
    Duration getTime()
    {
        endTime = std::chrono::high_resolution_clock::now();
        return std::chrono::duration_cast<Duration>(endTime - startTime);
    }

    template <class Duration = std::chrono::nanoseconds>
    Duration endTimer() {
        endTime = std::chrono::high_resolution_clock::now();
        return std::chrono::duration_cast<Duration>(endTime - startTime);
    }

    bool TLE() {
        return getTime() >= timeLimit;
    }
};


struct Grid {
    int cost;
    int prev;
    Grid() : cost(INT_MAX), prev(-1) {}
};

struct Pin {
    int x;
    int y;
    bool routed;
    Pin(int _x, int _y, int r) : x(_x), y(_y), routed(r) {}
};

void parse_basic(string filename, int &W, int &H, int &numOfPins) {
    ifstream fin(filename);

    // W, H
    fin >> W >> H;

    // # of pins
    fin >> numOfPins;
}

void parser(string filename, int W, int H, int numOfPins, int3 *pins, int *hWeight, int *vWeight, int2 *gridMap) {
    ifstream fin(filename);
    string str;
    
    // W, H
    fin >> W >> H;

    // # of pins
    fin >> numOfPins;

    // pins
    for (int i = 0; i < numOfPins; i++) {
        fin >> str;
        if (str != "Pin") cout << "Input: Pin error" << endl;

        int x, y;
        fin >> x >> y;
        pins[i].x = x;
        pins[i].y = y;
        pins[i].z = 0;
    }

    // vertical line weight
    for (int i = 0; i < H; i++) {
        fin >> str;
        if (str != "Vertical") cout << "Input: Vertical error" << endl;

        for (int j = 0; j < W - 1; j++) {
            fin >> vWeight[i * W + j];
        }
        vWeight[i * W + W - 1] = 0;
    }

    // horizontal line weight
    for (int i = 0; i < H - 1; i++) {
        fin >> str;
        if (str != "Horizontal") cout << "Input: Horizontal error" << endl;

        for (int j = 0; j < W; j++) {
            fin >> hWeight[i * W + j];
        }
    }
    // H
    for (int j = 0; j < W; j++) {
        hWeight[(H - 1) * W + j] = 0;
    }

    // grid map
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            gridMap[i * W + j].x = INT_MAX;
            gridMap[i * W + j].y = -1;
        }
    }

    // for testing
    /*
    cout << "Pins\n";
    for (int i = 0; i < numOfPins; i++) {
        cout << pins[i]->x << " " << pins[i]->y << endl;
    }
    cout << endl;

    cout << "vWeight\n";
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            cout << vWeight[i * W + j] << " ";
        }
        cout << endl;
    }
    cout << endl;

    cout << "hWeight\n";
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            cout << hWeight[i * W + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
    */
}

__global__ void initialize(int H, int W, int2 *gridMap) {
    int tid = threadIdx.x;
    int stride = blockDim.x;
    int i = blockIdx.x;

    if (i >= H) return ;

    for (int j = tid; j < W; j += stride) {
        if (gridMap[i * W + j].x != 0) {
            gridMap[i * W + j].x = INT_MAX;
        }
    }
}

void writeOutput(string filename, int H, int W, int2 *gridMap) {
    ofstream fout(filename);

    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            if (gridMap[i * W + j].x == 0) {
                fout << i << " " << j << "\n";
            }
        }
    }
    fout.close();
}

__device__ __host__ int transTo1D(int x, int y, int W) {
    return x * W + y;
}

__global__ void horizontalSweep1(int H, int W, int2 *gridMap, int *vWeight, int *changed) {
    int x = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];
    __shared__ int prefixSum[sharedSize];
    __shared__ int tmp[sharedSize];

    if (id >= W || x >= H) return ;

    for (int i = id; i < W; i += stride) {
        sharedGridMap[i] = gridMap[x * W + i];
        tmpWeight[i] = vWeight[x * W  + i];
    }

    __syncthreads();

    int offset = 1; 
    int n = sharedSize;

    for (int i = id; i < n; i += stride) {
        if (i >= W) prefixSum[i] = 0;
        else prefixSum[i] = tmpWeight[i];
    }

    __syncthreads();

	for (int d = n / 2; d > 0; d /= 2)	// build sum in place up the tree 
	{ 	
	    if (id < d) { 
        	int ai = offset * (2 * id + 1) - 1;
        	int bi = offset * (2 * id + 2) - 1;
        
            if (prefixSum[ai] != INT_MAX)
        	    prefixSum[bi] += prefixSum[ai];
        }
        
        offset *= 2;
	    __syncthreads();    
    }

    if (id == 0)
    {
    	prefixSum[n - 1] = 0;
    }	// clear the last element  

    for (int d = 1; d < n; d *= 2)	// traverse down tree &build scan 
    { 	
        offset /= 2;      
        __syncthreads();      
        if (id < d) { 

        	int ai = offset * (2 * id + 1) - 1;
            int bi = offset * (2 * id + 2) - 1;
            
            int t = prefixSum[ai];
            prefixSum[ai] = prefixSum[bi];
            if (t != INT_MAX)
                prefixSum[bi] += t;
        }
    }
    __syncthreads();

    for (int i = id; i < W; i += stride){
        tmp[i] = sharedGridMap[i].x - prefixSum[i];
    }

    __syncthreads();

    // if (id == 0) {
    //     for (int i = 1; i < W; i++) {
    //         if (tmp[i] > tmp[i - 1]) {
    //             tmp[i] = tmp[i - 1];
    //             sharedGridMap[i].y = transTo1D(x, i - 1, W);
    //             *changed = 1;
    //         }
    //     }
    // }

    int startIdx = 0;
    for (int i = id; i < W; i += stride) {
        if (i >= 1) {
            for (int s = startIdx; s < i; s++) {
                if (tmp[i] > tmp[s]) {
                    tmp[i] = tmp[s];
                    sharedGridMap[i].y = transTo1D(x, i - 1, W);
                    *changed = 1;
                }
            }
        }
        if (startIdx == 0) {
            startIdx = stride - 1;
        }
        else {
            startIdx += stride;
        }
        __syncthreads();
    }

    __syncthreads();

    for (int i = id; i < W; i += stride) {
        sharedGridMap[i].x = tmp[i] + prefixSum[i];
        gridMap[x * W + i] = sharedGridMap[i];
    }
}

__global__ void horizontalSweep2(int H, int W, int2 *gridMap, int *vWeight, int *changed) {
    int x = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];
    __shared__ int prefixSum[sharedSize];
    __shared__ int rprefixSum[sharedSize];
    __shared__ int tmp[sharedSize];

    if (id >= W || x >= H) return ;

    for (int i = id; i < W; i += stride) {
        sharedGridMap[i] = gridMap[x * W + i];
        tmpWeight[i] = vWeight[x * W  + i];
    }

    __syncthreads();

    int offset = 1; 
    int n = stride;

    for (int i = id; i < n; i += stride) {
        if (i >= W) prefixSum[i] = 0;
        else rprefixSum[i] = tmpWeight[W - i - 1];
    }

    __syncthreads();

	for (int d = n / 2; d > 0; d /= 2)	// build sum in place up the tree 
	{ 	
	    if (id < d) { 
        	int ai = offset * (2 * id + 1) - 1;
        	int bi = offset * (2 * id + 2) - 1;
        
            if (rprefixSum[ai] != INT_MAX)
        	    rprefixSum[bi] += rprefixSum[ai];
        }
        
        offset *= 2;
	    __syncthreads();    
    }

    if (id == 0)
    {
    	rprefixSum[n - 1] = 0;
    }	// clear the last element  

    for (int d = 1; d < n; d *= 2)	// traverse down tree &build scan 
    { 	
        offset /= 2;      
        __syncthreads();      
        if (id < d) { 

        	int ai = offset * (2 * id + 1) - 1;
            int bi = offset * (2 * id + 2) - 1;
            
            int t = rprefixSum[ai];
            rprefixSum[ai] = rprefixSum[bi];
            if (t != INT_MAX)
                rprefixSum[bi] += t;
        }
    }
    __syncthreads();

    for (int i = id; i < W; i += stride) {
        prefixSum[i] = rprefixSum[W - i - 1];
    }

    __syncthreads();

    for (int i = id; i < W; i += stride){
        tmp[i] = sharedGridMap[i].x - prefixSum[i];
    }

    __syncthreads();

    // if (id == 0) {
    //     for (int i = 1; i < W; i++) {
    //         if (tmp[i] > tmp[i - 1]) {
    //             tmp[i] = tmp[i - 1];
    //             sharedGridMap[i].y = transTo1D(x, i - 1, W);
    //             *changed = 1;
    //         }
    //     }
    // }

    int startIdx = W - 1;
    for (int i = id; i < W; i += stride) {
        int ci = W - i - 1;
        if (ci >= 1) {
            for (int s = startIdx; s > ci; s--) {
                if (tmp[ci] > tmp[s]) {
                    tmp[ci] = tmp[s];
                    sharedGridMap[ci].y = transTo1D(x, ci + 1, W);
                    *changed = 1;
                }
            }
        }
        if (startIdx == W - 1) {
            startIdx -= (stride - 1);
        }
        else {
            startIdx -= stride;
        }
        __syncthreads();
    }

    __syncthreads();

    for (int i = id; i < W; i += stride) {
        sharedGridMap[i].x = tmp[i] + prefixSum[i];
        gridMap[x * W + i] = sharedGridMap[i];
    }
}

__global__ void horizontalSweep3(int H, int W, int2 *gridMap, int *vWeight, int *changed) {
    int x = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];

    if (id >= W || x >= H) return ;

    for (int i = id; i < W; i += stride) {
        sharedGridMap[i] = gridMap[x * W + i];
        tmpWeight[i] = vWeight[x * W  + i];
    }

    __syncthreads();

    for (int i = 1; i <= ceil(log2f(W)); i++) {
        for (int j = id; j < W; j += stride) {
            int invert_j = W - 1 - j;
            if (invert_j % int(pow(2, i)) >= pow(2, i - 1)) {
                int idx = W - 1 - (invert_j - (invert_j % int(pow(2, i - 1))) - 1);
                int partialCost = 0;

                for (int k = idx - 1; k >= j; k--) {
                    partialCost += tmpWeight[k];
                }

                if (sharedGridMap[idx].x != INT_MAX && sharedGridMap[j].x > sharedGridMap[idx].x + partialCost) {
                    sharedGridMap[j].x = sharedGridMap[idx].x + partialCost;
                    sharedGridMap[j].y = transTo1D(x, j + 1, W);
                    *changed = 1;
                }
            }
        }
    }

    __syncthreads();

    for (int i = id; i < W; i += stride) {
        gridMap[x * W + i] = sharedGridMap[i];
    }
}

__global__ void verticalSweep1(int H, int W, int2 *gridMap, int *hWeight, int *changed) {
    int y = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];
    __shared__ int prefixSum[sharedSize];
    __shared__ int tmp[sharedSize];

    if (id >= H || y >= W) return ;

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i] = gridMap[i * W + y];
        tmpWeight[i] = hWeight[i * W + y];
    }

    __syncthreads();

    int offset = 1; 
    int n = stride;

    for (int i = id; i < n; i += stride) {
        if (i >= H) prefixSum[i] = 0;
        else prefixSum[i] = tmpWeight[i];
    }

    __syncthreads();

	for (int d = n / 2; d > 0; d /= 2)	// build sum in place up the tree 
	{ 	
	    if (id < d) { 
        	int ai = offset * (2 * id + 1) - 1;
        	int bi = offset * (2 * id + 2) - 1;
        
            if (prefixSum[ai] != INT_MAX)
        	    prefixSum[bi] += prefixSum[ai];
        }
        
        offset *= 2;
	    __syncthreads();    
    }

    if (id == 0)
    {
    	prefixSum[n - 1] = 0;
    }	// clear the last element  

    for (int d = 1; d < n; d *= 2)	// traverse down tree &build scan 
    { 	
        offset /= 2;      
        __syncthreads();      
        if (id < d) { 

        	int ai = offset * (2 * id + 1) - 1;
            int bi = offset * (2 * id + 2) - 1;
            
            int t = prefixSum[ai];
            prefixSum[ai] = prefixSum[bi];
            if (t != INT_MAX)
                prefixSum[bi] += t;
        }
    }
    __syncthreads();

    for (int i = id; i < H; i += stride){
        tmp[i] = sharedGridMap[i].x - prefixSum[i];
    }

    __syncthreads();

    // if (id == 0) {
    //     for (int i = 1; i < W; i++) {
    //         if (tmp[i] > tmp[i - 1]) {
    //             tmp[i] = tmp[i - 1];
    //             sharedGridMap[i].y = transTo1D(x, i - 1, W);
    //             *changed = 1;
    //         }
    //     }
    // }

    int startIdx = 0;
    for (int i = id; i < H; i += stride) {
        if (i >= 1) {
            for (int s = startIdx; s < i; s++) {
                if (tmp[i] > tmp[s]) {
                    tmp[i] = tmp[s];
                    sharedGridMap[i].y = transTo1D(i - 1, y, W);
                    *changed = 1;
                }
            }
        }
        if (startIdx == 0) {
            startIdx = stride - 1;
        }
        else {
            startIdx += stride;
        }
        __syncthreads();
    }

    __syncthreads();

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i].x = tmp[i] + prefixSum[i];
        gridMap[i * W + y] = sharedGridMap[i];
    }
}

__global__ void verticalSweep2(int H, int W, int2 *gridMap, int *hWeight, int *changed) {
    int y = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];
    __shared__ int prefixSum[sharedSize];
    __shared__ int rprefixSum[sharedSize];
    __shared__ int tmp[sharedSize];

    if (id >= H || y >= W) return ;

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i] = gridMap[i * W + y];
        tmpWeight[i] = hWeight[i * W  + y];
    }

    __syncthreads();

    int offset = 1; 
    int n = sharedSize;

    for (int i = id; i < n; i += stride) {
        if (i >= H) prefixSum[i] = 0;
        else rprefixSum[i] = tmpWeight[H - i - 1];
    }

    __syncthreads();

	for (int d = n / 2; d > 0; d /= 2)	// build sum in place up the tree 
	{ 	
	    if (id < d) { 
        	int ai = offset * (2 * id + 1) - 1;
        	int bi = offset * (2 * id + 2) - 1;
        
            if (rprefixSum[ai] != INT_MAX)
        	    rprefixSum[bi] += rprefixSum[ai];
        }
        
        offset *= 2;
	    __syncthreads();    
    }

    if (id == 0)
    {
    	rprefixSum[n - 1] = 0;
    }	// clear the last element  

    for (int d = 1; d < n; d *= 2)	// traverse down tree &build scan 
    { 	
        offset /= 2;      
        __syncthreads();      
        if (id < d) { 

        	int ai = offset * (2 * id + 1) - 1;
            int bi = offset * (2 * id + 2) - 1;
            
            int t = rprefixSum[ai];
            rprefixSum[ai] = rprefixSum[bi];
            if (t != INT_MAX)
                rprefixSum[bi] += t;
        }
    }
    __syncthreads();

    for (int i = id; i < H; i += stride) {
        prefixSum[i] = rprefixSum[H - i - 1];
    }

    __syncthreads();

    for (int i = id; i < H; i += stride){
        tmp[i] = sharedGridMap[i].x - prefixSum[i];
    }

    __syncthreads();

    // if (id == 0) {
    //     for (int i = 1; i < W; i++) {
    //         if (tmp[i] > tmp[i - 1]) {
    //             tmp[i] = tmp[i - 1];
    //             sharedGridMap[i].y = transTo1D(x, i - 1, W);
    //             *changed = 1;
    //         }
    //     }
    // }

    int startIdx = H - 1;
    for (int i = id; i < H; i += stride) {
        int ci = H - i - 1;
        if (ci >= 1) {
            for (int s = startIdx; s > ci; s--) {
                if (tmp[ci] > tmp[s]) {
                    tmp[ci] = tmp[s];
                    sharedGridMap[ci].y = transTo1D(ci + 1, y, W);
                    *changed = 1;
                }
            }
        }
        if (startIdx == H - 1) {
            startIdx -= (stride - 1);
        }
        else {
            startIdx -= stride;
        }
        __syncthreads();
    }

    __syncthreads();

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i].x = tmp[i] + prefixSum[i];
        gridMap[i * W + y] = sharedGridMap[i];
    }
}

__global__ void verticalSweep3(int H, int W, int2 *gridMap, int *hWeight, int *changed) {
    int y = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];

    if (id >= H || y >= W) return ;

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i] = gridMap[i * W + y];
        tmpWeight[i] = hWeight[i * W + y];
    }

    __syncthreads();

    for (int i = 1; i <= ceil(log2f(H)); i++) {
        for (int j = id; j < H; j += stride) {
            if (j % int(pow(2, i)) >= pow(2, i - 1)) {
                int idx = j - (j % int(pow(2, i - 1))) - 1;
                int partialCost = 0;

                for (int k = idx; k < j; k++) {
                    partialCost += tmpWeight[k];
                }

                if (sharedGridMap[idx].x != INT_MAX && sharedGridMap[j].x > sharedGridMap[idx].x + partialCost) {
                    sharedGridMap[j].x = sharedGridMap[idx].x + partialCost;
                    sharedGridMap[j].y = transTo1D(j - 1, y, W);
                    *changed = 1;
                }
            }
        }
    }

    __syncthreads();

    for (int i = id; i < H; i += stride) {
        gridMap[i * W + y] = sharedGridMap[i];
    }    
}

__global__ void verticalSweep4(int H, int W, int2 *gridMap, int *hWeight, int *changed) {
    int y = blockIdx.x;
    int id = threadIdx.x;
    int stride = blockDim.x;

    __shared__ int2 sharedGridMap[sharedSize];
    __shared__ int tmpWeight[sharedSize];

    if (id >= H || y >= W) return ;

    for (int i = id; i < H; i += stride) {
        sharedGridMap[i] = gridMap[i * W + y];
        tmpWeight[i] = hWeight[i * W  + y];
    }

    __syncthreads();

    for (int i = 1; i <= ceil(log2f(H)); i++) {
        for (int j = id; j < H; j += stride) {
            int invert_j = H - 1 - j;
            if (invert_j % int(pow(2, i)) >= pow(2, i - 1)) {
                int idx = H - 1 - (invert_j - (invert_j % int(pow(2, i - 1))) - 1);
                int partialCost = 0;

                for (int k = idx - 1; k >= j; k--) {
                    partialCost += tmpWeight[k];
                }

                if (sharedGridMap[idx].x != INT_MAX && sharedGridMap[j].x > sharedGridMap[idx].x + partialCost) {
                    sharedGridMap[j].x = sharedGridMap[idx].x + partialCost;
                    sharedGridMap[j].y = transTo1D(j + 1, y, W);
                    *changed = 1;
                }
            }
        }
    }

    __syncthreads();

    for (int i = id; i < H; i += stride) {
        gridMap[i * W + y] = sharedGridMap[i];
    }    
}

__global__ void getMinCostPin(int H, int W, int numOfPins, int2 *gridMap, int3 *pins) {
    int tid = threadIdx.x;
    int minPin = -1;
    // int minCost = INT_MAX;

    if (blockIdx.x != 0) return ;

    __shared__ int minPins[NumOfThreads];
    for (int i = threadIdx.x; i < NumOfThreads; i += blockDim.x) {
        if (i < numOfPins) minPins[i] = i;
        else minPins[i] = 0;
    }

    __syncthreads();

    for (int size = NumOfThreads / 2; size > 0; size /= 2) {
        if (tid < size) {
            int t = minPins[tid];
            int cmp = minPins[tid + size];
            if (pins[cmp].z == 0) {
                if (pins[t].z == 1 || gridMap[pins[cmp].x * W + pins[cmp].y].x < gridMap[pins[t].x * W + pins[t].y].x) {
                    minPins[tid] = cmp;
                }
            }
        }

        __syncthreads();
    }

    if (tid == 0) {
        minPin = minPins[0];
        // printf("%d\n", minPin);

        // retrace the path & set the cost = 0
        int x = pins[minPin].x;
        int y = pins[minPin].y;

        while (gridMap[x * W + y].x != 0) {
            // for debug
            if (gridMap[x * W + y].y == -1) {
                printf("getMinCost: cannot retrace the path\n");
                // cout << "getMinCost: cannot retrace the path" << endl;
            }

            gridMap[x * W + y].x = 0;
            int p = gridMap[x * W + y].y;
            x = p / W;
            y = p % W;
        }
        pins[minPin].z = 1;
    }


    // for testing 
    /*
    cout << "Pin: " << minPin << endl;
    cout << "Coord: " << pins[minPin]->x << " " << pins[minPin]->y << endl;
    cout << endl;
    */
}


// for testing
void printGridMap(int H, int W, int2 *gridMap) {
    for (int i = 0; i < H; i++) {
        for (int j = 0; j < W; j++) {
            if (gridMap[i * W + j].x == INT_MAX) 
                cout << setw(3) << "INF" << " ";
            else 
                cout << setw(3) << gridMap[i * W + j].x << " ";
        }
        cout << endl;
    }
    cout << endl;
}

// void sweep() {
//     int iteration = 0;
//     while (true) {
//         iteration++;
//         changed = false;
//         horizontalSweep();
//         verticalSweep();
        
//         if (!changed) break;
//     }

//     cout << "Iteration: " << iteration << endl;
// }

int main(int argc, char **argv) {
    // check the argument is enough
    if (argc != 3) {
        cout << "The number of arguments are wrong!!" << endl;
        return 0;
    }

    int W, H, numOfPins;
    int *hWeight, *vWeight;
    int2 *gridMap;
    int *changed;
    int3 *pins;

    globalTimer::startTimer();
    // parse basic
    parse_basic(argv[1], W, H, numOfPins);

    // new 
    hWeight = new int [W * H];
    vWeight = new int [W * H];
    gridMap = new int2 [W * H];
    changed = new int;
    pins = new int3 [numOfPins];

    // parse the input file
    parser(argv[1], W, H, numOfPins, pins, hWeight, vWeight, gridMap);
    std::chrono::nanoseconds _input = globalTimer::endTimer();

    // routing (sweep)
    gridMap[pins[0].x * W + pins[0].y].x = 0;
    pins[0].z = 1;

    // for testing 
    // printGridMap();

    // cuda
    const int numOfThreads = NumOfThreads;
    const int numOfBlocks = W;

    // cudaMalloc
    int *dhWeight, *dvWeight;
    int2 *dgridMap;
    int *dchanged;
    int3 *dpins;

    hipMalloc(&dhWeight, H * W * sizeof(int));
    hipMalloc(&dvWeight, H * W * sizeof(int));
    hipMalloc(&dgridMap, H * W * sizeof(int2));
    hipMalloc(&dpins, numOfPins * sizeof(int3));
    hipMalloc(&dchanged, sizeof(int));

    hipMemcpy(dhWeight, hWeight, H * W * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dvWeight, vWeight, H * W * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dgridMap, gridMap, H * W * sizeof(int2), hipMemcpyHostToDevice);
    hipMemcpy(dpins, pins, numOfPins * sizeof(int3), hipMemcpyHostToDevice);
    hipMemcpy(dchanged, changed, sizeof(int), hipMemcpyHostToDevice);

    globalTimer::startTimer();
    for (int i = 0; i < numOfPins - 1; i++) {
        initialize<<<numOfBlocks, numOfThreads>>>(H, W, dgridMap);
        hipDeviceSynchronize();

        int iteration = 0;
        while (true) {
            iteration++;
            *changed = 0;
            hipMemcpy(dchanged, changed, sizeof(int), hipMemcpyHostToDevice);

            horizontalSweep1<<<numOfBlocks, numOfThreads>>>(H, W, dgridMap, dvWeight, dchanged);
            hipDeviceSynchronize();
            horizontalSweep2<<<numOfBlocks, numOfThreads>>>(H, W, dgridMap, dvWeight, dchanged);
            hipDeviceSynchronize();
            verticalSweep3<<<numOfBlocks, numOfThreads>>>(H, W, dgridMap, dhWeight, dchanged);
            hipDeviceSynchronize();
            verticalSweep4<<<numOfBlocks, numOfThreads>>>(H, W, dgridMap, dhWeight, dchanged);
            hipDeviceSynchronize();

            hipMemcpy(gridMap, dgridMap, H * W * sizeof(int2), hipMemcpyDeviceToHost);
            printGridMap(H, W, gridMap);
            
            hipMemcpy(changed, dchanged, sizeof(int), hipMemcpyDeviceToHost);
            if (*changed == 0) break;
        }

        // cout << "Iteration: " << iteration << endl;

        // sweep();
        getMinCostPin<<<numOfBlocks, numOfThreads>>>(H, W, numOfPins, dgridMap, dpins);

        // for testing 
        // printGridMap();
    }
    std::chrono::nanoseconds _kernel = globalTimer::endTimer();

    globalTimer::startTimer();
    hipMemcpy(gridMap, dgridMap, H * W * sizeof(int2), hipMemcpyDeviceToHost);
    writeOutput(argv[2], H, W, gridMap);
    std::chrono::nanoseconds _output = globalTimer::endTimer();

    // cout time usage
    auto inputTime = _input.count();
    auto outputTime = _output.count();
    auto kernelTime = _kernel.count();

    // cout << "I/O time: " << inputTime / 1e9 + outputTime / 1e9 << endl;
    cout << "Kernel time: " << kernelTime / 1e9 << endl;

    // delete 
    delete []hWeight;
    delete []vWeight;
    delete []gridMap;
    delete []changed;
    delete []pins;

    // cudaFree
    hipFree(dhWeight);
    hipFree(dvWeight);
    hipFree(dgridMap);
    hipFree(dchanged);
    hipFree(dpins);

    return 0;
}